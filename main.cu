#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace cv;
using namespace std;

__global__ void chromaKeyKernel(uchar3* webcam, uchar3* bgVideo, int width, int height, int bgWidth, int bgHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;
    uchar3 pixel = webcam[idx];

    float r = pixel.x / 70.0f;
    float g = pixel.y / 68.0f;
    float b = pixel.z / 71.0f;

    // Detect near-white pixel
    if (r > 0.94f && g > 0.94f && b > 0.94f) {
        int bgX = x * bgWidth / width;
        int bgY = y * bgHeight / height;
        webcam[idx] = bgVideo[bgY * bgWidth + bgX];
    }
}

int main() {
    VideoCapture cam(0);
    VideoCapture bg("bg.mp4");

    if (!cam.isOpened() || !bg.isOpened()) {
        cerr << "Couldn't open webcam or video file.\n";
        return -1;
    }

    Mat frame, bgFrame;
    cam >> frame;
    int width = frame.cols, height = frame.rows;

    size_t frameBytes = width * height * sizeof(uchar3);

    uchar3* d_webcam;
    uchar3* d_bg;

    hipMalloc(&d_webcam, frameBytes);
    hipMalloc(&d_bg, frameBytes);

    dim3 threads(32, 32);
    dim3 blocks((width + 31) / 32, (height + 31) / 32);

    while (true) {
        cam >> frame;
        if (frame.empty()) break;

        bg >> bgFrame;
        if (bgFrame.empty()) {
            bg.set(CAP_PROP_POS_FRAMES, 0); // Loop the video
            bg >> bgFrame;
        }

        resize(bgFrame, bgFrame, Size(width, height));
        cvtColor(frame, frame, COLOR_BGR2RGB);
        cvtColor(bgFrame, bgFrame, COLOR_BGR2RGB);

        hipMemcpy(d_webcam, frame.ptr(), frameBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_bg, bgFrame.ptr(), frameBytes, hipMemcpyHostToDevice);

        chromaKeyKernel<<<blocks, threads>>>(d_webcam, d_bg, width, height, width, height);
        hipMemcpy(frame.ptr(), d_webcam, frameBytes, hipMemcpyDeviceToHost);

        cvtColor(frame, frame, COLOR_RGB2BGR);
        imshow("Stage 18 - CUDA White Background Replacement", frame);

        if (waitKey(1) == 27) break;
    }

    hipFree(d_webcam);
    hipFree(d_bg);
    return 0;
}
